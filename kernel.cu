#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""



__global__ void addIndex(int* a, int* b, int* c){
    //index, der fortæller hvilken thread vi executer kode i
    //hver thread executer en blok
    int i = threadIdx.x; 
    c[i] = a[i] + b[i]; 
}


void main(){

    //memory på vores host
    const int count = 5; 
    int ha[] = {1, 2, 3, 4, 5}; 
    int hb[] = {10, 20, 30, 40, 50}; 
    int hc[count]; 

    //den plads vi skal bruge på gpu'en til at execute koden for da, db og dc
    const size = count * sizeof(int);
    int *da, *db, *dc; 
    hipMalloc(&da, size); 
    hipMalloc(&db, size); 
    hipMalloc(&dc, size); 

    //kopier koden fra ha til da osv
    hipMemcpy(da, ha, size, hipMemcpyKind:: hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyKind:: hipMemcpyHostToDevice);
    hipMemcpy(dc, hc, size, hipMemcpyKind:: hipMemcpyHostToDevice);



    //simulere processen 
    for(int i = 0; i <count; ++i){
        addIndex(a, b, c); 
    }


}