#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h> 

#define GPU_RUNS 100

void squareSerial(float* d_in, float* d_out, int N){
    for (unsigned int i = 0; i < N; ++i){
        d_out[i] = pow(d_in[i]/(d_in[i]-2.3), 3);
    }
}

__global__ void squareKernel(float* d_in, float* d_out, int N){
    const unsigned int lid = threadIdx.x; 
    const unsigned int gid = blockIdx.x*blockDim.x + lid; 
    if(gid < N){
        d_out[gid] = pow(d_in[gid]/(d_in[gid]-2.3), 3);
    }
}


int timeval_substract(struct timeval* result, struct timeval* t2, struct timeval* t1){
    unsigned int resolution = 1000000; 
    long int diff = (t2 -> tv_usec + resolution * t2 -> tv_sec) - (t1 -> tv_usec + resolution * t1 -> tv_sec);
    result -> tv_sec = diff/resolution; 
    result -> tv_usec = diff % resolution; 
    return (diff <0); 
}

int main(int argc, char** argv){
    unsigned int N = 753411; //størrelsen på arrayet
    unsigned int mem_size = N*sizeof(float); //størrelsen på hukommelsen der skal bruges til arrayet
    unsigned int block_size = 256; //størrelsen på en block
    unsigned int num_blocks = ((N + (block_size -1))/block_size); //antallet af blocks


    //For measure the time 
    unsigned long int elaped; struct timeval t_start, t_end, t_diff; 
    gettimeofday(t_start, NULL); 

    //allocates host-memory
    float* h_in = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    //initialize the memory
    for(unsigned int i = 0; i <N; ++i){
            h_in[i] = float(i);
    }

    //allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in, mem_size);
    hipMalloc((void**)&d_out, mem_size);

    //copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    //execute the kernel and calculates the square using gpu 
    for(int i = 0; i < GPU_RUNS; i++){
        squareKernel <<<num_blocks, block_size>>>(d_in, d_out, N);
    }hipDeviceSynchronize(); 

    gettimeofday(&t_diff, NULL); 
    timeval_substract(&t_diff, &t_end, &t_start); 
    elaped = (t_diff.tv_sec*1e6+t_diff.tv_usec)/GPU_RUNS; 
    printf("Took %d microseconds (%.2fms)\n", elaped, elaped/1000.0);

    //copy result from device to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    //print result
    //for(unsigned int i = 0; i <N; i++) printf("%.6f\n", h_out[i]);

    //Calculates squareSerial using the cpu
    float* cpu_res = (float*) malloc(mem_size);
    squareSerial(h_in, cpu_res, N); 

    //Checks the results are the same
     
    for (unsigned int j = 0; j < N; ++j){
        if(fabs(cpu_res[j] - h_out[j]) < 0.0001){
            printf("VALID \n");
        }else{
            printf("INVALID\n");
        }
    }
    

    //mål tiden 
    //undersøg hvornår gpuen bliver hurtigere 


    //clean-up memory
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

}