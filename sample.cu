#include <stdlib.h>
#include <stdio.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


//tager to arrays som input
__global__ void squareKernel(float* d_in, float* d_out, int N){
    const unsigned int lid = threadIdx.x; //forstår ikke hvad linjen gør
    const unsigned int gid = blockIdx.x*blockDim.x + lid; //forstår ikke hvad linjen gør
    d_out[gid] = d_in[gid]*d_in[gid];
}

int main(int argc, char** argv){
    unsigned int N = 32; //længden af arrayet
    unsigned int mem_size = N*sizeof(float); //hvor meget hukommelse vi skal bruge

    //allocates host-memory
    float* h_in = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    //initialize the memory
    for(unsigned int i = 0; i <N; ++i){
            h_in[i] = float(i);
    }

    //allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in, mem_size);
    hipMalloc((void**)&d_out, mem_size);

    //copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    //execute the kernel
    squareKernel <<<1, N>>>(d_in, d_out, N);

    //copy result from device to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    //print result
    for(unsigned int i = 0; i <N; i++) printf("%.6f\n", h_out[i]);

    //clean-up memory
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

}